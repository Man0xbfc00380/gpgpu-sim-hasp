/* file: hello.cu */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main()
{
    int *c;
    c = (int*)malloc(sizeof(int));
    int *dev_c;
    hipMalloc((void **)&dev_c, sizeof(int));
    add<<<1, 1>>>(2, 7, dev_c);
    hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("[cuda] 2 + 7 = %d\n",*c);
    return 0;
}