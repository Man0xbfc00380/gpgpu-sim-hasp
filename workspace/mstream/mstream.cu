#include <stdio.h>
#include <hip/hip_runtime.h>

__global__  void  haspSet_vectorAddKernel_th0_sh10_mem8(){}

__global__  void  haspSet_vectorMulKernel_th1_sh10_mem8(){}

__global__  void  vectorAddKernel(int* a,  int* b,  int* c,  int N) {
    int  tid;
    tid  =  threadIdx.x;
    if (tid < N) c[tid] = a[tid] + b[tid];
}

__global__  void  vectorMulKernel(int* a,  int* b,  int* c,  int N) {
    int  tid;
    tid  =  threadIdx.x;
    if (tid < N) c[tid] = a[tid] * b[tid];
}

int main()
{
    int N = 1000;

    int* h_a = new int[N];
    int* h_b = new int[N];
    int* h_c = new int[N];
    int* h_d = new int[N];

    for (int i = 0; i < N; i++) {
        h_a[i] = 1 + i;
        h_b[i] = 2;
    }

    int *d_a, *d_b, *d_c, *d_d;
    
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));
    hipMalloc(&d_d, N * sizeof(int));

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 32;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    haspSet_vectorAddKernel_th0_sh10_mem8<<<blocksPerGrid, threadsPerBlock, 0, stream1>>> ();
    haspSet_vectorMulKernel_th1_sh10_mem8<<<blocksPerGrid, threadsPerBlock, 0, stream2>>> ();

    vectorAddKernel<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_a, d_b, d_c, N);
    hipMemcpyAsync(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost, stream1);

    vectorMulKernel<<<blocksPerGrid, threadsPerBlock, 0, stream2>>>(d_a, d_b, d_d, N);
    hipMemcpyAsync(h_d, d_d, N * sizeof(int), hipMemcpyDeviceToHost, stream2);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    for (int i = 0; i < 10; i++) {
        printf("id: %d, (i+1) + 2 = %d, (i+1) * 2 = %d\n", i, h_c[i], h_d[i]);
    }

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    delete[] h_d;
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);

    return 0;
}

