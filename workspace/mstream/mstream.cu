#include <stdio.h>
#include <hip/hip_runtime.h>

__global__  void  haspSet_vectorAddKernel_th1_sh5_mem4(){}
__global__  void  haspSet_vectorMulKernel_th2_sh25_mem20(){}
__global__  void  vectorAddKernel(int* a,  int* b,  int* c,  int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) c[tid] = a[tid] + b[tid];
}
__global__  void  vectorMulKernel(int* a,  int* b,  int* c,  int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) c[tid] = a[tid] * b[tid];
}

int main()
{
    int N  = 500000;
    int N1 = 100000;
    int N2 = 500000;

    int* h_a = new int[N];
    int* h_b = new int[N];
    int* h_c = new int[N];
    int* h_d = new int[N];

    for (int i = 0; i < N; i++) {
        h_a[i] = 1 + i;
        h_b[i] = 2;
    }

    int *d_a, *d_b, *d_c, *d_d;
    
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));
    hipMalloc(&d_d, N * sizeof(int));

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 64;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("[Grid: %d, Block: %d]\n", blocksPerGrid, threadsPerBlock);
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    haspSet_vectorAddKernel_th1_sh5_mem4<<<1, 1, 0, stream1>>> ();
    haspSet_vectorMulKernel_th2_sh25_mem20<<<1, 1, 0, stream2>>> ();

    vectorAddKernel<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_a, d_b, d_c, N1);
    hipMemcpyAsync(h_c, d_c, N * sizeof(int), hipMemcpyDeviceToHost, stream1);

    vectorMulKernel<<<blocksPerGrid, threadsPerBlock, 0, stream2>>>(d_a, d_b, d_d, N2);
    hipMemcpyAsync(h_d, d_d, N * sizeof(int), hipMemcpyDeviceToHost, stream2);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);

    for (int i = 0; i < 5; i++) {
        printf("id: %d, (i+1) + 2 = %d, (i+1) * 2 = %d\n", i, h_c[i], h_d[i]);
    }
    printf("... ...\n");
    for (int i = 995; i < 1000; i++) {
        printf("id: %d, (i+1) + 2 = %d, (i+1) * 2 = %d\n", i, h_c[i], h_d[i]);
    }

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    delete[] h_d;

    return 0;
}

