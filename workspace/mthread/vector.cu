#include  "vector.h"
#include  "vector_kernel.h"

void  vectorAdd( float   * a,  float   * b,  float   * c,  int  n) {
    float   * d_a,  * d_b,  * d_c;
    haspSet_vectorAddKernel_th0_sh10_mem8<<< 1, 1 >>> ();
    hipMalloc(( void   ** ) & d_a, n  *   sizeof ( float ));
    hipMemcpy(d_a, a, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    hipMalloc(( void   ** ) & d_b, n  *   sizeof ( float ));
    hipMemcpy(d_b, b, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    hipMalloc(( void   ** ) & d_c, n  *   sizeof ( float ));
    hipMemcpy(d_b, b, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    vectorAddKernel <<< 1 , n >>> (d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, n  *   sizeof ( float ), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void  vectorSub( float   * a,  float   * b,  float   * c,  int  n) {
    float   * d_a,  * d_b,  * d_c;
    haspSet_vectorSubKernel_th1_sh10_mem8 <<< 1, 1 >>> ();
    hipMalloc(( void   ** ) & d_a, n  *   sizeof ( float ));
    hipMemcpy(d_a, a, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    hipMalloc(( void   ** ) & d_b, n  *   sizeof ( float ));
    hipMemcpy(d_b, b, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    hipMalloc(( void   ** ) & d_c, n  *   sizeof ( float ));
    hipMemcpy(d_b, b, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    vectorSubKernel <<< 1 , n >>> (d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, n  *   sizeof ( float ), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void  vectorMul( float   * a,  float   * b,  float   * c,  int  n) {
    float   * d_a,  * d_b,  * d_c;
    haspSet_vectorMulKernel_th1_sh10_mem8 <<< 1, 1 >>> ();
    hipMalloc(( void   ** ) & d_a, n  *   sizeof ( float ));
    hipMemcpy(d_a, a, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    hipMalloc(( void   ** ) & d_b, n  *   sizeof ( float ));
    hipMemcpy(d_b, b, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    hipMalloc(( void   ** ) & d_c, n  *   sizeof ( float ));
    hipMemcpy(d_b, b, n  *   sizeof ( float ), hipMemcpyHostToDevice);
    vectorMulKernel <<< 1 , n >>> (d_a, d_b, d_c, n);
    hipMemcpy(c, d_c, n  *   sizeof ( float ), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}